#include "hip/hip_runtime.h"
#include "header.h"

#include <stdio.h>
#include <hip/hip_runtime.h>
#include <omp.h>
#include <iostream>
#include <cstdlib>

#include "adabs/vector.h"
#include "adabs/cuda_device/accessors.h"

__device__ void down (int *temp) {
  int offset = 1;

  for (int d = (blockDim.x*2)>>1; d>0; d >>= 1) {
    __syncthreads();
    if (threadIdx.x < d) {
      const int ai = offset * (2*threadIdx.x+1) - 1;
      const int bi = offset * (2*threadIdx.x+2) - 1;
      temp[bi] += temp[ai];
    }
    offset *= 2;
  }
}

__device__ void up (int *temp) {
  int offset = blockDim.x*2;
  
  for (int d=1; d<(blockDim.x*2); d *= 2) {
    offset >>= 1;
    __syncthreads();
    if (threadIdx.x < d) {
      const int ai = offset * (2*threadIdx.x+1) - 1;
      const int bi = offset * (2*threadIdx.x+2) - 1;
      const int t = temp[ai];
      temp[ai]  = temp[bi];
      temp[bi] += t;
    }
  }
}

__global__ void block_scan (int* input, int* output, int* block_sum) {
	volatile const int* reader = get_tile(input, blockIdx.x, 0, blockDim.x*2);
	
	__shared__ int temp[256];
	temp[2*threadIdx.x]   = reader[2*threadIdx.x];
	temp[2*threadIdx.x+1] = reader[2*threadIdx.x+1];
	
	down(temp);
	__syncthreads();
	
	int* block_sum_writer = get_tile_unitialized(block_sum, blockIdx.x, 0, 1);
	if (threadIdx.x == 0) {
		block_sum_writer[0] = temp[255];
		temp[255] = 0;
	}
	
	__syncthreads();
	set_tile(block_sum, blockIdx.x, 0, 1);
	
	up(temp);
	
	int* writer = get_tile_unitialized(output, blockIdx.x, 0, blockDim.x*2);
	writer[2*threadIdx.x]   = temp[2*threadIdx.x];
	writer[2*threadIdx.x+1] = temp[2*threadIdx.x+1];
	set_tile(output, blockIdx.x, 0, blockDim.x*2);
}


__global__ void block_add (int* input, int* output, int* block_sum_scanned) {
	volatile const int* reader = get_tile(input, blockIdx.x+1, 0, blockDim.x*2);
	volatile const int* block_sum_reader = get_tile(block_sum_scanned, blockIdx.x+1, 0, 1);
	
	__shared__ int temp[256];
	temp[2*threadIdx.x]   = reader[2*threadIdx.x];
	temp[2*threadIdx.x+1] = reader[2*threadIdx.x+1];
	
	int add = block_sum_reader[0];
	 
	temp[2*threadIdx.x]   += add;
	temp[2*threadIdx.x+1] += add;
	
	int* writer = get_tile_unitialized(output, blockIdx.x+1, 0, blockDim.x*2);
	writer[2*threadIdx.x]   = temp[2*threadIdx.x];
	writer[2*threadIdx.x+1] = temp[2*threadIdx.x+1];
	set_tile(output, blockIdx.x+1, 0, blockDim.x*2);
}

/*inline double timediff(timeval tv2, timeval tv1) {
	return (double) (tv2.tv_sec - tv1.tv_sec) + ((double) (tv2.tv_usec - tv1.tv_usec) / 1000000.0);
}*/

void caller() {
	timeval tv1, tv2, tv3, tv4;
	
	hipError_t error;
    int nb_of_blocks = 1400;
	std::cout << "#blocks: " << nb_of_blocks << std::endl;
	
	const int block_size = 256;
	adabs::vector< adabs::cuda_host::local <int> > input(nb_of_blocks*block_size, block_size);
	adabs::vector< adabs::cuda_host::local <int> > output_block_scan(nb_of_blocks*block_size, block_size);
	adabs::vector< adabs::cuda_host::local <int> > output_final(nb_of_blocks*block_size, block_size);

	adabs::vector< adabs::cuda_host::local <int> > block_sums(nb_of_blocks, 1);
	adabs::vector< adabs::cuda_host::local <int> > block_sums_scanned(nb_of_blocks, 1);

	hipSetDevice(0);
	// start block scan kernel @ GPU 0
	block_scan<<<nb_of_blocks, block_size/2>>>((int*)input.get_distri().get_data_addr().get_raw_pointer(),
		                                       (int*)output_block_scan.get_distri().get_data_addr().get_raw_pointer(),
		                                       (int*)block_sums.get_distri().get_data_addr().get_raw_pointer()
		                                      );

	hipSetDevice(1);
	// start add kernel @ GPU 1
	block_add<<<nb_of_blocks-1, block_size/2>>>((int*)output_block_scan.get_distri().get_data_addr().get_raw_pointer(),
		                                        (int*)output_final.get_distri().get_data_addr().get_raw_pointer(),
		                                        (int*)block_sums_scanned.get_distri().get_data_addr().get_raw_pointer()
		                                       );
	omp_set_num_threads(4);
	
	gettimeofday(&tv1, NULL);
	#pragma omp parallel
	{
		int me = omp_get_thread_num();
		int all = omp_get_num_threads();

		// fill input array with random numbers
		#pragma omp single nowait
		{
			gettimeofday(&tv3, NULL);
			for (int i=0; i<nb_of_blocks; ++i) {
				int *writer = input.get_unitialized(i*block_size);
				for (int j=0; j<block_size; ++j) {
					writer[j] = std::rand() % 256;//block_size - j;
				}
				
				volatile int x = 0;
				for (int X=0; X<50000; ++X)
					x+=X;
				
				input.set(i*block_size, writer);
			}
			gettimeofday(&tv4, NULL);
		}

		// scan block sums

		#pragma omp single nowait
		{
			int start = 0;
			for (int i=0; i<nb_of_blocks; ++i) {
				int *writer = block_sums_scanned.get_unitialized(i);
				writer[0] = start;
				block_sums_scanned.set(i, writer);
			
				const int reader = block_sums.get(i);
				start += reader;
			}
		}
	
		/*// copy block 0
		#pragma omp single nowait
		{
			const int *reader = output_block_scan.get_tile(0);
			int *writer = output_final.get_unitialized(0);
			for (int j=0; j<block_size; ++j) {
				writer[j] = reader[j];
			}
			output_final.set(0, writer);
		}

		// read final results + sanity check (may crash due to overflow)
		#pragma omp single nowait
		{
			int prev = -1;
			for (int i=0; i<nb_of_blocks; ++i) {
				const int *reader = output_final.get_tile(i*block_size);
			
				for (int j=0; j<block_size; ++j) {
					if (prev > reader[j]) {
						const int *reader2 = input.get_tile(i*block_size);
						const int *reader3 = output_block_scan.get_tile(i*block_size);
						const int reader4 = block_sums_scanned.get(i);
						for (int j=0; j<block_size; ++j) {
							std::cout << i*block_size+j << " - " << reader2[j] << " - " << reader3[j] << " - " << reader4 << " - " << reader[j] << std::endl;
						}
					}
					assert (prev <= reader[j]);
					prev = reader[j];
				}
			}
		}*/

	}

	gettimeofday(&tv2, NULL);
	std::cout << "runtime: " << timediff(tv2, tv1) << std::endl;
	std::cout << "fill time: " << timediff(tv4, tv3) << std::endl;
	

	// make sure our next call to hipGetLastError will return errors
	// from the kernels started before
	hipDeviceSynchronize();

	// check for error
	error = hipGetLastError();
	if(error != hipSuccess) {
		// print the CUDA error message and exit
		printf("CUDA error: %s\n", hipGetErrorString(error));
		exit(-1);
	}
}

#if 0
__global__ void test(float *A, float *B) {
	float* writer = get_tile_unitialized(A, blockIdx.x, blockIdx.y, blockDim.x*blockDim.y);
	volatile const float* reader = get_tile(B, blockIdx.x, blockIdx.y, blockDim.x*blockDim.y);
	
	writer[threadIdx.x+threadIdx.y*blockDim.x] = reader[threadIdx.x+threadIdx.y*blockDim.x];
	__syncthreads();
	 
	set_tile(A, blockIdx.x, blockIdx.y, blockDim.x*blockDim.y);
}
#endif
	#if 0
void caller(/*adabs::matrix< adabs::cuda_host::local < float > > &A,
            adabs::matrix< adabs::cuda_host::local < float > > &B,
            adabs::matrix< adabs::cuda_host::local < float > > &C*/) {
   	dim3 block_d(A.get_distri().get_batch_size_x(), A.get_distri().get_batch_size_y());
	dim3 grid_d(A.get_distri().get_size_x() / block_d.x, A.get_distri().get_size_y() / block_d.y);
	
	test <<< grid_d, block_d >>> ((float*)A.get_distri().get_data_addr().get_raw_pointer(), (float*)B.get_distri().get_data_addr().get_raw_pointer());

	for (int i=0; i<grid_d.x; ++i) {
		for (int j=0; j<grid_d.y; ++j) {
			float *b_ptr = B.get_tile_unitialized(i*block_d.x, j*block_d.y);
			for (int ii=0; ii<block_d.y; ++ii) {
				for (int jj=0; jj<block_d.x; ++jj) {
					b_ptr[ii*block_d.x + jj] = jj; 
				}
			}
			B.set_tile(i*block_d.x, j*block_d.y, b_ptr);
		}
	}

	
	for (int i=0; i<grid_d.x; ++i) {
		for (int j=0; j<grid_d.y; ++j) {
			const float *a_ptr = A.get_tile(i*block_d.x, j*block_d.y);
			for (int ii=0; ii<block_d.y; ++ii) {
				for (int jj=0; jj<block_d.x; ++jj) {
					if (a_ptr[ii*block_d.x + jj] != jj) {
						for (int x = jj; x<jj+10 && x<block_d.x; ++x)
							std::cout << "(" << i << ", " << j << ", " << ii << ", " << x << ") - " << a_ptr[ii*block_d.x + x] << std::endl; 
					}
					assert (a_ptr[ii*block_d.x + jj] == jj);
				}
			}
		}
	}
	
	#endif

